#ifndef CUDA_GENERAL_CU
#define CUDA_GENERAL_CU

/** A file of common CUDA includes and simple functions **/
/** to be directly #included by other cuda files.       **/

#include <hipblas.h>

#include "General.h"

// Define to check for errors after each cuda call
//#define DEBUG_CUDA

#ifdef DEBUG_CUDA
#define CHECKCUDA(s) {hipError_t err = hipDeviceSynchronize(); \
                      if( err != hipSuccess ) { \
                        cerr << "CUDA " << s \
                             << ": " << hipGetErrorString(err) << endl; \
                        exit(1); \
                      }}
#else
#define CHECKCUDA(s)
#endif

#define SAFECUDA(s) s;CHECKCUDA(#s)


// A quick class to time gpu kernels using device events
struct StopWatch_GPU
{
  hipEvent_t startTime, stopTime;
  StopWatch_GPU()  { hipEventCreate(&startTime); hipEventCreate(&stopTime); }
  ~StopWatch_GPU() { hipEventDestroy(startTime); hipEventDestroy(stopTime); }
  inline void start() { hipEventRecord(startTime,0); }
  inline double stop() { return elapsed(); }
  inline double elapsed() 
  {
    hipEventRecord(stopTime,0);
    hipEventSynchronize(stopTime);
    float result;
    hipEventElapsedTime(&result, startTime, stopTime);
    return result/1000.0;    // 1000 mSec per Sec
  }
};





inline int cudaMaxSMEM()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  return deviceProp.sharedMemPerBlock;
}


template <class T>
inline void cudaMemcpyD2D( T* d_a, const T* d_b, int N )
{
  DEBUG_TOTAL(StopWatch_GPU timer;  timer.start());
  SAFECUDA(hipMemcpy( d_a, d_b, N*sizeof(T), hipMemcpyDeviceToDevice ))
  //CHECKCUDA("MemcpyD2D Error");
  INCR_TOTAL(Transfer,timer.stop());
  //cout << "D2D: " << N << endl;
}

template <class T>
inline void cudaMemcpyH2D( T* d_a, const T* h_a, int N )
{
  DEBUG_TOTAL(StopWatch_GPU timer;  timer.start());
  hipMemcpy( d_a, h_a, N*sizeof(T), hipMemcpyHostToDevice );
  CHECKCUDA("MemcpyH2D Error");
  INCR_TOTAL(Transfer,timer.stop());
  //cout << "H2D: " << N << endl;
}

template <class T>
inline void cudaMemcpyD2H( T* h_a, const T* d_a, int N )
{
  DEBUG_TOTAL(StopWatch_GPU timer;  timer.start());
  hipMemcpy( h_a, d_a, N*sizeof(T), hipMemcpyDeviceToHost );
  CHECKCUDA("MemcpyD2H Error");
  INCR_TOTAL(Transfer,timer.stop());
  //cout << "D2H: " << N << endl;
}



template <class T>
inline void cudaMemcpyH2D( T* d_a, const vector<T>& h_a )
{
  cudaMemcpyH2D( d_a, &h_a[0], h_a.size() );
}

template <class T>
inline void cudaMemcpyD2H( vector<T>& h_a, const T* d_a )
{
  cudaMemcpyD2H( &h_a[0], d_a, h_a.size() );
}



inline void cudaDelete( void* d_a )
{
  hipFree( d_a );
  CHECKCUDA("hipFree Error");
}



template <class T>
inline T* cudaNew( int N, const T* h_a = NULL )
{
  T* d_a = NULL;

  hipMalloc( (void**)&d_a, N*sizeof(T) );
  CHECKCUDA("Malloc Error");

  if( h_a != NULL )
    cudaMemcpyH2D( d_a, h_a, N );

  return d_a;
}

template <class T>
inline T* cudaNew( const vector<T>& h_a )
{
  return cudaNew( h_a.size(), &h_a[0] );
}


inline void cudaInit(int device = 0)
{
  StopWatch initTimer;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if( deviceCount == 0 ) {
    cout << "Error: No devices supporting CUDA" << endl;
    exit(1);
  }
  
  if( device < 0 )               device = 0;
  if( device > deviceCount - 1 ) device = deviceCount - 1;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  if (deviceProp.major < 1) {
    cerr << "Error: " << deviceProp.name << " does not support CUDA." << endl;
    exit(1);
  }

  hipSetDevice(device);
  cerr << "Initializing " << deviceProp.name << "... ";
  
  int* temp = cudaNew<int>(1);
  cudaDelete( temp );

  cerr << initTimer.stop() << "s" << endl << endl;
}












#endif
